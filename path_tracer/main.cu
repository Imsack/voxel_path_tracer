#pragma once

#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>
#include <surface_functions.h>
#include <iostream>

#include "render_data.cuh"
#include "scene_generation.cuh"
#include "game_loop.cuh"
#include "path_tracing.cuh"


uint32_t screen_GL_texture;
hipGraphicsResource_t screen_cuda_resource;

void render(G_Buffer& g_buffer, Scene& scene, Player player)
{
    hipGraphicsMapResources(1, &screen_cuda_resource);

    hipArray_t screen_cuda_array;
    hipGraphicsSubResourceGetMappedArray(&screen_cuda_array, screen_cuda_resource, 0, 0);

    hipResourceDesc screen_cuda_array_resource_desc = {};
    screen_cuda_array_resource_desc.resType = hipResourceTypeArray;
    screen_cuda_array_resource_desc.res.array.array = screen_cuda_array;

    hipSurfaceObject_t screen_cuda_surface_object;
    hipCreateSurfaceObject(&screen_cuda_surface_object, &screen_cuda_array_resource_desc);

    // render here

    dim3 grid(20, 45, 1);
    dim3 block(32, 8, 1);


    path_tracing<<<grid, block>>>(screen_cuda_surface_object, g_buffer, scene, player);

    regulate_probes<<<grid, block>>>(g_buffer, scene, player);

    render_frame<<<grid, block>>>(screen_cuda_surface_object, g_buffer, scene);


    hipDestroySurfaceObject(screen_cuda_surface_object);

    hipGraphicsUnmapResources(1, &screen_cuda_resource);

    hipStreamSynchronize(0);

    glBindTexture(GL_TEXTURE_2D, screen_GL_texture);

    glBegin(GL_QUADS);

    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(+1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(+1.0f, +1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, +1.0f);

    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);

    glFinish();
}

int main()
{
    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(SCREEN_W, SCREEN_H, "Path Tracer", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    glfwSwapInterval(0);

    // copied initialization code
    glEnable(GL_TEXTURE_2D);

    glGenTextures(1, &screen_GL_texture);

    glBindTexture(GL_TEXTURE_2D, screen_GL_texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, SCREEN_W, SCREEN_H, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glBindTexture(GL_TEXTURE_2D, 0);

    hipGraphicsGLRegisterImage(&screen_cuda_resource, screen_GL_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);


    // setup render data

    Player player({ 0.1f, 1.5f, 0.1f }, 0, 0);

    G_Buffer g_buffer = G_Buffer();

    Scene scene;

    generate_scene(scene);


    float time = 0;
    int frame_count = 0;

    float time_step = 0;

    while (!glfwWindowShouldClose(window))
    {
        double initial_time = glfwGetTime();

        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        input(scene, player, time_step, window);
        // simulate(scene, player, time_step);
        render(g_buffer, scene, player);

        ++player.frame_id;

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();


        double final_time = glfwGetTime();

        time_step = final_time - initial_time;

        time += time_step;
        ++frame_count;

        if (time > 1.0)
        {
            std::cout << "FPS: " << frame_count << '\n';
            std::cout << "ms/frame: " << 1000.0 / frame_count << '\n';

            time = 0;
            frame_count = 0;
        }
    }

    glfwTerminate();

    return 0;
}